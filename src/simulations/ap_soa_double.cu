#include "hip/hip_runtime.h"
#include "utilities/time_utils.h"
#include "utilities/nbody_helpers.h"
#include <iostream>
#include <omp.h>

#define MAX_THREADS_PER_BLOCK 1024
#include <cstring> // Add this line to include the necessary header for strcmp
#include <cstdlib> // Add this line to include the necessary header for exit
#include <algorithm>

/**
 * Sets the CUDA device based on the provided device name.
 *
 * @param deviceName The name of the device to set.
 */
void setDeviceByName(const char* deviceName) {
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  int targetDevice = -1;
  for (int i = 0; i < deviceCount; ++i) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, i);
    std::string fullName(deviceProp.name);
    std::string shortName(deviceName);
    // Convert both names to lowercase for case-insensitive comparison
    std::transform(fullName.begin(), fullName.end(), fullName.begin(), ::tolower);
    std::transform(shortName.begin(), shortName.end(), shortName.begin(), ::tolower);
    if (fullName.find(shortName) != std::string::npos) {
      targetDevice = i;
      break;
    }
  }
  if (targetDevice != -1) {
    hipSetDevice(targetDevice);
    std::cout << "Device " << deviceName << " set as the current device." << std::endl;
  } else {
    std::cerr << "Device " << deviceName << " not found." << std::endl;
    exit(1);
  }
}

double inline xoxyi_rand(unsigned int *seed){
  return (double)rand_r(seed) / (double) RAND_MAX;
}

/**
 * Calculates the potential energy (Epot) of a system of particles using the Lennard-Jones potential.
 *
 * @param n The number of particles in the system.
 * @param p Pointer to an array of double4 structures representing the particles.
 *          Each double4 structure contains the x, y, z coordinates of a particle (p[i].x, p[i].y, p[i].z)
 *          and the weight of the particle (p[i].w).
 * @return The total potential energy of the system.
 */
double inline Ep(const int n, double4 *p) {
  double Epot = 0.0f;
#pragma omp parallel for reduction(+:Epot)
  for (int i = 0; i < n; ++i) {
    for (int j = i + 1; j < n; ++j) {
      double D, x, y, z;
      x = p[i].x - p[j].x;
      y = p[i].y - p[j].y;
      z = p[i].z - p[j].z;
      D = sqrt(x * x + y * y + z * z);
      Epot += -1.0f * p[i].w * p[j].w / D;
    }
  }
  return Epot;
}

/**
 * Calculates the kinetic energy of a system of particles.
 *
 * @param n The number of particles in the system.
 * @param v Pointer to an array of double4 structures representing the particles' properties.
 *           Each double4 structure contains the particle's position (x, y, z) and mass (w).
 * @return The total kinetic energy of the system.
 */
double inline Ek(const int n, double4 *v) {
  double Ekin = 0.0;
#pragma omp parallel for reduction(+:Ekin)
  for (int i = 0; i < n; ++i) {
    Ekin += 0.5f * v[i].w * (v[i].x * v[i].x + v[i].y * v[i].y + v[i].z * v[i].z);
  }
  return Ekin;
}

/**
 * Scales a 4D array by a given scale factor.
 *
 * @param n     The size of the array.
 * @param m     The 4D array to be scaled.
 * @param scale The scale factor to multiply each element of the array by.
 */
static inline void scale4DArray(const int n, double4 *m, const double scale) {
#pragma omp parallel for 
  for (int i = 0; i < n; ++i) {
    m[i].x *= scale;
    m[i].y *= scale;
    m[i].z *= scale;
  }
}

/**
 * @brief Initializes the position and velocity arrays for a given number of particles.
 * 
 * @param n The number of particles.
 * @param p Pointer to the position array.
 * @param v Pointer to the velocity array.
 * @param seed The seed value for the random number generator.
 */
void InitPosVel(const int n, double4 *p, double4 *v, int seed) {
#pragma omp parallel
  {
    unsigned int seedT = omp_get_thread_num() + seed * omp_get_num_threads();	 
    #pragma omp for
    for (int i = 0; i < n; ++i) {
      double R, X, Y;
      double mi = _M / n;
      R = xoxyi_rand(&seedT);
      X = acos(1.0f - 2.0f * xoxyi_rand(&seedT));
      Y = xoxyi_rand(&seedT) * 2.0f * _PI;
      // https://www.researchgate.net/figure/Figure-A1-Spherical-coordinates_fig8_284609648
      p[i] = make_double4(R * sin(X) * cosf(Y), R * sinf(X) * sinf(Y), R * cosf(X), mi);
      v[i] = make_double4(1.0f - 2.0f * xoxyi_rand(&seedT), 1.0f - 2.0f * xoxyi_rand(&seedT), 1.0f - 2.0f * xoxyi_rand(&seedT), mi);
    }
  }
}

/**
 * Moves the particles to the center of mass and subtracts the center of mass velocity from each particle.
 *
 * @param n The number of particles.
 * @param p Pointer to an array of double4 structures representing the position of each particle.
 * @param v Pointer to an array of double4 structures representing the velocity of each particle.
 */
void Move2Center(const int n, double4 *p, double4 *v) {
  double3 pp = make_double3(0.0f, 0.0f, 0.0f);
  double3 vv = make_double3(0.0f, 0.0f, 0.0f);
  double ppx = 0;
  double ppy = 0;
  double ppz = 0;
  double vvx = 0;
  double vvy = 0;
  double vvz = 0;
  int i;

  // Calculate the total position and velocity of all particles
#pragma omp parallel for reduction(+:ppx,ppy,ppz,vvx,vvy,vvz)
  for (i = 0; i < n; ++i) {
    ppx += p[i].x * p[i].w;
    ppy += p[i].y * p[i].w;
    ppz += p[i].z * p[i].w;

    vvx += v[i].x * v[i].w;
    vvy += v[i].y * v[i].w;
    vvz += v[i].z * v[i].w;
  }

  // Calculate the center of mass position and velocity
  pp.x = ppx;
  pp.y = ppy;
  pp.z = ppz;

  vv.x = vvx;
  vv.y = vvy;
  vv.z = vvz;

  pp.x /= _M;
  pp.y /= _M;
  pp.z /= _M;
  vv.x /= _M;
  vv.y /= _M;
  vv.z /= _M;

  // Move particles to the center of mass and subtract the center of mass velocity
#pragma omp parallel for
  for (i = 0; i < n; ++i) {
    p[i].x -= pp.x;
    p[i].y -= pp.y;
    p[i].z -= pp.z;
    v[i].x -= vv.x;
    v[i].y -= vv.y;
    v[i].z -= vv.z;
  }
}

/**
 * Rescales the energy of the particles in the simulation.
 *
 * This function implements Algorithm 7.2 from Aarseth, 2003.
 * It rescales the potential energy and kinetic energy of the particles
 * to achieve a desired virial ratio and energy balance.
 *
 * @param n The number of particles.
 * @param p Pointer to the array of position vectors.
 * @param v Pointer to the array of velocity vectors.
 */
void RescaleEnergy(const int n, double4 *p, double4 *v) {
  // Aarseth, 2003, Algorithm 7.2.
  double Epot = Ep(n, p);
  double Ekin = Ek(n, v);
  double virialRatio = 0.5f;
  double Qv = sqrt(virialRatio * abs(Epot) / Ekin);
  scale4DArray(n, v, Qv);
  double beta = abs((1 - virialRatio) * Epot / (Epot + Ekin));

  scale4DArray(n, p, beta);
  scale4DArray(n, v, 1.0f / (sqrt(beta)));

  // After first scale Ekin is -0.5Epot but E0 != -0.25.
  // So just scale up or down as needed.
  Epot = Ep(n, p);
  beta = Epot / -0.5f;
  scale4DArray(n, p, beta);
  scale4DArray(n, v, 1.0f / sqrt(beta));
}

/**
 * Initializes the positions, velocities, and masses of the bodies.
 *
 * @param n The number of bodies.
 * @param p Pointer to the array of body positions.
 * @param v Pointer to the array of body velocities.
 * @param seed The seed for the random number generator (default is 0).
 */
void InitBodies(const int n, double4 *p, double4 *v, int seed = 0) {
  // Initialize masses equally
  InitPosVel(n, p, v, seed);

  // Translate bodies to move the center of mass on center of the coordinate
  // system
  Move2Center(n, p, v);

  // Rescale energy
  RescaleEnergy(n, p, v);
}

/**
 * @brief Computes the interactions between particles using the Barnes-Hut algorithm.
 *
 * This CUDA kernel calculates the forces between particles in a simulation using the Barnes-Hut algorithm.
 * It iterates over each particle and computes the forces between that particle and all other particles.
 * The forces are then used to update the velocities of the particles.
 *
 * @param n The number of particles in the simulation.
 * @param p Pointer to an array of double4 structures representing the positions and masses of the particles.
 * @param v Pointer to an array of double4 structures representing the velocities of the particles.
 * @param dt The time step for the simulation.
 */
__global__ void ComputeInteractions(const int n, double4 *p, double4 *v, const double dt) {
  double fx = 0.0f;
  double fy = 0.0f;
  double fz = 0.0f;
  auto i = blockDim.x * blockIdx.x + threadIdx.x;
  #pragma unroll
  for (int j = 0; j < n; ++j) {
      // compute distance pair
      auto dx = p[j].x - p[i].x;
      auto dy = p[j].y - p[i].y;
      auto dz = p[j].z - p[i].z;

      auto d = dx * dx + dy * dy + dz * dz + _SOFTENING*_SOFTENING;
      auto d_inv = 1.0f / sqrt(d);
      auto d_inv3 = d_inv * d_inv * d_inv;

      fx += d_inv3 * p[j].w * dx;
      fy += d_inv3 * p[j].w * dy;
      fz += d_inv3 * p[j].w * dz;  
  }
  v[i].x += fx * dt;
  v[i].y += fy * dt;
  v[i].z += fz * dt;
}

/**
 * @brief Updates the position of particles in a simulation.
 *
 * This CUDA kernel function updates the position of particles based on their velocity and the given time step.
 *
 * @param p - Pointer to an array of double4 structures representing the positions of particles.
 * @param v - Pointer to an array of double4 structures representing the velocities of particles.
 * @param dt - The time step used for the update.
 */
__global__ void UpdatePosition(double4 *p, double4 *v, const double dt) {
  auto i = blockDim.x * blockIdx.x + threadIdx.x;
  p[i].x += v[i].x * dt;
  p[i].y += v[i].y * dt;
  p[i].z += v[i].z * dt;
}


int main (int argc, char **argv) {
  int seed = 0;
  if (argc < 2) {
    std::cerr << "Must specify the number of bodies" << std::endl;
    exit(1);
  }
  if (argc == 3)
    seed = atoi(argv[2]);
  else  
    srand(seed);

  int n = atoi(argv[1]);
  const double dt = 0.01f; 
  
  double4 *h_p, *h_v;

  // Set device specified as argument to the program
  setDeviceByName(argv[3]);

  // Allocate pinned memory
  hipHostMalloc(&h_p, n * sizeof(double4));
  hipHostMalloc(&h_v, n * sizeof(double4));

  // Init Bodies
  TIMERSTART(init)
  InitBodies(n, h_p, h_v, seed);
#ifdef MONITOR_ENERGY
  {
  double ek = Ek(n, h_v);
  double ep = Ep(n, h_p);
  std::cout << "Etot: " <<ek+ep <<std::endl;
  }
#endif
  TIMERSTOP(init)

  // Allocate memory on the device
  double4 *d_p, *d_v;
  hipMalloc(&d_p, n * sizeof(double4));
  hipMalloc(&d_v, n * sizeof(double4));

  dim3 blocks, threadsPerBlock;
  if (n < MAX_THREADS_PER_BLOCK) {
    blocks = dim3(1);
    threadsPerBlock = dim3(n);
  } else {
    // assuming that n is a power of two
    blocks = dim3(n / MAX_THREADS_PER_BLOCK);
    threadsPerBlock = dim3(MAX_THREADS_PER_BLOCK);
  }

  TIMERSTART(total)
  hipMemcpy(d_p, h_p, n * sizeof(double4), hipMemcpyHostToDevice);
  hipMemcpy(d_v, h_v, n * sizeof(double4), hipMemcpyHostToDevice);
  
  TIMERSTART(simulation)
  for (double t = 0; t < 100; t+= dt) {
    ComputeInteractions<<<blocks, threadsPerBlock>>>(n, d_p, d_v, dt );
    UpdatePosition<<<blocks, threadsPerBlock>>>(d_p, d_v, dt);
    hipMemcpy(h_p, d_p, n * sizeof(double4), hipMemcpyDeviceToHost);
    hipMemcpy(h_v, d_v, n * sizeof(double4), hipMemcpyDeviceToHost);
#ifdef MONITOR_ENERGY
    {
      double ek = Ek(n, h_v);
      double ep = Ep(n, h_p);
      std::cout << "Etot: " <<ek+ep <<std::endl;
    }
#endif
  }
  hipDeviceSynchronize();
  TIMERSTOP(simulation)

  //hipMemcpy(h_p, d_p, n * sizeof(double4), hipMemcpyDeviceToHost);
  //hipMemcpy(h_v, d_v, n * sizeof(double4), hipMemcpyDeviceToHost);
  TIMERSTOP(total)

#ifdef MONITOR_ENERGY
  {
    double ek = Ek(n, h_v);
    double ep = Ep(n, h_p);
    std::cout << "Etot: " <<ek+ep <<std::endl;
  }
#endif
  
  // Free pinned memory
  hipHostFree(h_p);
  hipHostFree(h_v);

  // Free memory on device
  hipFree(d_p);
  hipFree(d_v);

  return 0;
}
