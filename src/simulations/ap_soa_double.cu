#include "hip/hip_runtime.h"
#include "utilities/time_utils.h"
#include "utilities/nbody_helpers.h"
#include <iostream>
#include <omp.h>

#define MAX_THREADS_PER_BLOCK 1024
#include <cstring> // Add this line to include the necessary header for strcmp
#include <cstdlib> // Add this line to include the necessary header for exit
#include <algorithm>

void setDeviceByName(const char* deviceName) {
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  int targetDevice = -1;
  for (int i = 0; i < deviceCount; ++i) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, i);
    std::string fullName(deviceProp.name);
    std::string shortName(deviceName);
    // Convert both names to lowercase for case-insensitive comparison
    std::transform(fullName.begin(), fullName.end(), fullName.begin(), ::tolower);
    std::transform(shortName.begin(), shortName.end(), shortName.begin(), ::tolower);
    if (fullName.find(shortName) != std::string::npos) {
      targetDevice = i;
      break;
    }
  }
  if (targetDevice != -1) {
    hipSetDevice(targetDevice);
    std::cout << "Device " << deviceName << " set as the current device." << std::endl;
  } else {
    std::cerr << "Device " << deviceName << " not found." << std::endl;
    exit(1);
  }
}

double inline xoxyi_rand(unsigned int *seed){
  return (double)rand_r(seed) / (double) RAND_MAX;
}

double inline Ep(const int n, double4 *p) {
  double Epot = 0.0f;
#pragma omp parallel for reduction(+:Epot)
  for (int i = 0; i < n; ++i) {
    for (int j = i + 1; j < n; ++j) {
      double D, x, y, z;
      x = p[i].x - p[j].x;
      y = p[i].y - p[j].y;
      z = p[i].z - p[j].z;
      D = sqrt(x * x + y * y + z * z);
      Epot += -1.0f * p[i].w * p[j].w / D;
    }
  }
  return Epot;
}

double inline Ek(const int n, double4 *v) {
  double Ekin = 0.0;
#pragma omp parallel for reduction(+:Ekin)
  for (int i = 0; i < n; ++i) {
    Ekin += 0.5f * v[i].w * (v[i].x * v[i].x + v[i].y * v[i].y + v[i].z * v[i].z);
  }
  return Ekin;
}

static inline void scale4DArray(const int n, double4 *m, const double scale) {
#pragma omp parallel for 
  for (int i = 0; i < n; ++i) {
    m[i].x *= scale;
    m[i].y *= scale;
    m[i].z *= scale;
  }
}

void InitPosVel(const int n, double4 *p, double4 *v) {
#pragma omp parallel
  {
    unsigned int seed = omp_get_thread_num();	 
    #pragma omp for
    for (int i = 0; i < n; ++i) {
      double R, X, Y;
      double mi = _M / n;
      R = xoxyi_rand(&seed);
      X = acos(1.0f - 2.0f * xoxyi_rand(&seed));
      Y = xoxyi_rand(&seed) * 2.0f * _PI;
      // https://www.researchgate.net/figure/Figure-A1-Spherical-coordinates_fig8_284609648
      p[i] = make_double4(R * sin(X) * cosf(Y), R * sinf(X) * sinf(Y), R * cosf(X), mi);
      v[i] = make_double4(1.0f - 2.0f * xoxyi_rand(&seed), 1.0f - 2.0f * xoxyi_rand(&seed), 1.0f - 2.0f * xoxyi_rand(&seed), mi);
    }
  }
}

void Move2Center(const int n, double4 *p, double4 *v) {
  double3 pp = make_double3(0.0f, 0.0f, 0.0f);
  double3 vv = make_double3(0.0f, 0.0f, 0.0f);
  double ppx = 0;
  double ppy = 0;
  double ppz = 0;
  double vvx = 0;
  double vvy = 0;
  double vvz = 0;
  int i;
#pragma omp parallel for reduction(+:ppx,ppy,ppz,vvx,vvy,vvz)
  for (i = 0; i < n; ++i) {
    ppx += p[i].x * p[i].w;
    ppy += p[i].y * p[i].w;
    ppz += p[i].z * p[i].w;

    vvx += v[i].x * v[i].w;
    vvy += v[i].y * v[i].w;
    vvz += v[i].z * v[i].w;
  }
  pp.x = ppx;
  pp.y = ppy;
  pp.z = ppz;

  vv.x = vvx;
  vv.y = vvy;
  vv.z = vvz;

  pp.x /= _M;
  pp.y /= _M;
  pp.z /= _M;
  vv.x /= _M;
  vv.y /= _M;
  vv.z /= _M;

#pragma omp parallel for
  for (i = 0; i < n; ++i) {
    p[i].x -= pp.x;
    p[i].y -= pp.y;
    p[i].z -= pp.z;
    v[i].x -= vv.x;
    v[i].y -= vv.y;
    v[i].z -= vv.z;
  }
}

void RescaleEnergy(const int n, double4 *p, double4 *v) {
  // Aarseth, 2003, Algorithm 7.2.
  double Epot = Ep(n, p);
  double Ekin = Ek(n, v);
  double virialRatio = 0.5f;
  double Qv = sqrt(virialRatio * abs(Epot) / Ekin);
  scale4DArray(n, v, Qv);
  double beta = abs((1 - virialRatio) * Epot / (Epot + Ekin));

  scale4DArray(n, p, beta);
  scale4DArray(n, v, 1.0f / (sqrt(beta)));

  // After first scale Ekin is -0.5Epot but E0 != -0.25.
  // So just scale up or down as needed.
  Epot = Ep(n, p);
  beta = Epot / -0.5f;
  scale4DArray(n, p, beta);
  scale4DArray(n, v, 1.0f / sqrt(beta));
}

void InitBodies(const int n, double4 *p, double4 *v) {
  // Initialize masses equally
  InitPosVel(n, p, v);

  // Translate bodies to move the center of mass on center of the coordinate
  // system
  Move2Center(n, p, v);

  // Rescale energy
  RescaleEnergy(n, p, v);
}

__global__ void ComputeInteractions(const int n, double4 *p, double4 *v, const double dt) {
  double fx = 0.0f;
  double fy = 0.0f;
  double fz = 0.0f;
  auto i = blockDim.x * blockIdx.x + threadIdx.x;
  #pragma unroll
  for (int j = 0; j < n; ++j) {
      // compute distance pair
      auto dx = p[j].x - p[i].x;
      auto dy = p[j].y - p[i].y;
      auto dz = p[j].z - p[i].z;

      auto d = dx * dx + dy * dy + dz * dz + _SOFTENING*_SOFTENING;
      auto d_inv = 1.0f / sqrt(d);
      auto d_inv3 = d_inv * d_inv * d_inv;

      fx += d_inv3 * p[j].w * dx;
      fy += d_inv3 * p[j].w * dy;
      fz += d_inv3 * p[j].w * dz;  
  }
  v[i].x += fx * dt;
  v[i].y += fy * dt;
  v[i].z += fz * dt;
}

__global__ void UpdatePosition(double4 *p, double4 *v, const double dt) {
  auto i = blockDim.x * blockIdx.x + threadIdx.x;
  p[i].x += v[i].x * dt;
  p[i].y += v[i].y * dt;
  p[i].z += v[i].z * dt;
}


int main (int argc, char **argv) {
  if (argc < 2) {
    std::cerr << "Must specify the number of bodies" << std::endl;
    exit(1);
  }
  if (argc == 3)
    srand(atoi(argv[2]));
  else  
    srand(0);

  int n = atoi(argv[1]);
  const double dt = 0.01f; 
  
  double4 *h_p, *h_v;

  // Set device specified as argument to the program
  setDeviceByName(argv[3]);

  // Allocate pinned memory
  hipHostMalloc(&h_p, n * sizeof(double4));
  hipHostMalloc(&h_v, n * sizeof(double4));

  // Init Bodies
  TIMERSTART(init)
  InitBodies(n, h_p, h_v);
  {
  double ek = Ek(n, h_v);
  double ep = Ep(n, h_p);
  std::cout << "Etot: " <<ek+ep <<std::endl;
  }
  TIMERSTOP(init)

  // Allocate memory on the device
  double4 *d_p, *d_v;
  hipMalloc(&d_p, n * sizeof(double4));
  hipMalloc(&d_v, n * sizeof(double4));

  dim3 blocks, threadsPerBlock;
  if (n < MAX_THREADS_PER_BLOCK) {
    blocks = dim3(1);
    threadsPerBlock = dim3(n);
  } else {
    // assuming that n is a power of two
    blocks = dim3(n / MAX_THREADS_PER_BLOCK);
    threadsPerBlock = dim3(MAX_THREADS_PER_BLOCK);
  }

  TIMERSTART(total)
  hipMemcpy(d_p, h_p, n * sizeof(double4), hipMemcpyHostToDevice);
  hipMemcpy(d_v, h_v, n * sizeof(double4), hipMemcpyHostToDevice);
  
  TIMERSTART(simulation)
  for (double t = 0; t < 0.1; t+= dt) {
    ComputeInteractions<<<blocks, threadsPerBlock>>>(n, d_p, d_v, dt );
    UpdatePosition<<<blocks, threadsPerBlock>>>(d_p, d_v, dt);
  }
  hipDeviceSynchronize();
  TIMERSTOP(simulation)

  hipMemcpy(h_p, d_p, n * sizeof(double4), hipMemcpyDeviceToHost);
  
  hipMemcpy(h_v, d_v, n * sizeof(double4), hipMemcpyDeviceToHost);
  TIMERSTOP(total)

  {
  double ek = Ek(n, h_v);
  double ep = Ep(n, h_p);
  std::cout << "Etot: " <<ek+ep <<std::endl;
  }
  
  // Free pinned memory
  hipHostFree(h_p);
  hipHostFree(h_v);

  // Free memory on device
  hipFree(d_p);
  hipFree(d_v);

  return 0;
}
