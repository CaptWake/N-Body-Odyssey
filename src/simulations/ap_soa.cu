#include "hip/hip_runtime.h"
#include "utilities/time_utils.h"
#include "utilities/nbody_helpers.h"
#include <iostream>

#define MAX_THREADS_PER_BLOCK 32

float inline Ep(const int n, float4 *p) {
  float Epot = 0.0f;
  float D, x, y, z;
  for (int i = 0; i < n; ++i) {
    for (int j = i + 1; j < n; ++j) {
      x = p[i].x - p[j].x;
      y = p[i].y - p[j].y;
      z = p[i].z - p[j].z;
      D = sqrtf(x * x + y * y + z * z);
      Epot += -1.0f * p[i].w * p[j].w / D;
    }
  }
  return Epot;
}

float inline Ek(const int n, float4 *v) {
  float Ekin = 0.0;
  for (int i = 0; i < n; ++i) {
    Ekin += 0.5f * v[i].w * (v[i].x * v[i].x + v[i].y * v[i].y + v[i].z * v[i].z);
  }
  return Ekin;
}

static inline void scale4DArray(const int n, float4 *m, const float scale) {
  for (int i = 0; i < n; ++i) {
    m[i].x *= scale;
    m[i].y *= scale;
    m[i].z *= scale;
  }
}

void InitPosVel(const int n, float4 *p, float4 *v) {
  float R, X, Y;
  float mi = _M / n;
  for (int i = 0; i < n; ++i) {
    R = fdrand<float>();
    X = acosf(1.0f - 2.0f * fdrand<float>());
    Y = fdrand<float>() * 2.0f * _PI;
    // https://www.researchgate.net/figure/Figure-A1-Spherical-coordinates_fig8_284609648
    p[i] = make_float4(R * sinf(X) * cosf(Y), R * sinf(X) * sinf(Y), R * cosf(X), mi);
    v[i] = make_float4(1.0f - 2.0f * fdrand<float>(), 1.0f - 2.0f * fdrand<float>(), 1.0f - 2.0f * fdrand<float>(), mi);
  }
}

void Move2Center(const int n, float4 *p, float4 *v) {
  float3 pp = make_float3(0.0f, 0.0f, 0.0f);
  float3 vv = make_float3(0.0f, 0.0f, 0.0f);
  int i;
  for (i = 0; i < n; ++i) {
    pp.x += p[i].x * p[i].w;
    pp.y += p[i].y * p[i].w;
    pp.z += p[i].z * p[i].w;

    vv.x += v[i].x * v[i].w;
    vv.y += v[i].y * v[i].w;
    vv.z += v[i].z * v[i].w;
  }

  pp.x /= _M;
  pp.y /= _M;
  pp.z /= _M;
  vv.x /= _M;
  vv.y /= _M;
  vv.z /= _M;

  for (i = 0; i < n; ++i) {
    p[i].x -= pp.x;
    p[i].y -= pp.y;
    p[i].z -= pp.z;
    v[i].x -= vv.x;
    v[i].y -= vv.y;
    v[i].z -= vv.z;
  }
}

void RescaleEnergy(const int n, float4 *p, float4 *v) {
  // Aarseth, 2003, Algorithm 7.2.
  float Epot = Ep(n, p);
  float Ekin = Ek(n, v);
  float virialRatio = 0.5f;
  float Qv = sqrtf(virialRatio * fabsf(Epot) / Ekin);
  scale4DArray(n, v, Qv);
  float beta = fabsf((1 - virialRatio) * Epot / (Epot + Ekin));

  scale4DArray(n, p, beta);
  scale4DArray(n, v, 1.0f / (sqrtf(beta)));

  // After first scale Ekin is -0.5Epot but E0 != -0.25.
  // So just scale up or down as needed.
  Epot = Ep(n, p);
  beta = Epot / -0.5f;
  scale4DArray(n, p, beta);
  scale4DArray(n, v, 1.0f / sqrtf(beta));
}

void InitBodies(const int n, float4 *p, float4 *v) {
  // Initialize masses equally
  InitPosVel(n, p, v);

  // Translate bodies to move the center of mass on center of the coordinate
  // system
  Move2Center(n, p, v);

  // Rescale energy
  RescaleEnergy(n, p, v);
}

__global__ void ComputeInteractions(const int n, float4 *p, float4 *v, const float dt) {
  float fx = 0.0f;
  float fy = 0.0f;
  float fz = 0.0f;
  auto i = blockDim.x * blockIdx.x + threadIdx.x;
  #pragma unroll
  for (int j = 0; j < n; ++j) {
      // compute distance pair
      auto dx = p[j].x - p[i].x;
      auto dy = p[j].y - p[i].y;
      auto dz = p[j].z - p[i].z;

      auto d = dx * dx + dy * dy + dz * dz + _SOFTENING*_SOFTENING;
      auto d_inv = 1.0f / sqrtf(d);
      auto d_inv3 = d_inv * d_inv * d_inv;

      fx += d_inv3 * p[j].w * dx;
      fy += d_inv3 * p[j].w * dy;
      fz += d_inv3 * p[j].w * dz;  
  }
  v[i].x += fx * dt;
  v[i].y += fy * dt;
  v[i].z += fz * dt;
}

__global__ void UpdatePosition(float4 *p, float4 *v, const float dt) {
  auto i = blockDim.x * blockIdx.x + threadIdx.x;
  p[i].x += v[i].x * dt;
  p[i].y += v[i].y * dt;
  p[i].z += v[i].z * dt;
}


int main (int argc, char **argv) {
  if (argc < 2) {
    std::cerr << "Must specify the number of bodies" << std::endl;
    exit(1);
  }
  if (argc == 3)
    srand(atoi(argv[2]));
  else  
    srand(0);
  
  int n = atoi(argv[1]);
  const float dt = 0.01f; 
  
  float4 *h_p, *h_v;

  // Allocate pinned memory
  hipHostMalloc(&h_p, n * sizeof(float4));
  hipHostMalloc(&h_v, n * sizeof(float4));

  // Init Bodies
  InitBodies(n, h_p, h_v);

  // Allocate memory on the device
  float4 *d_p, *d_v;
  hipMalloc(&d_p, n * sizeof(float4));
  hipMalloc(&d_v, n * sizeof(float4));

  dim3 blocks, threadsPerBlock;
  if (n < MAX_THREADS_PER_BLOCK) {
    blocks = dim3(1);
    threadsPerBlock = dim3(n);
  } else {
    // assuming that n is a power of two
    blocks = dim3(n / MAX_THREADS_PER_BLOCK);
    threadsPerBlock = dim3(MAX_THREADS_PER_BLOCK);
  }

  TIMERSTART(total)
  hipMemcpy(d_p, h_p, n * sizeof(float4), hipMemcpyHostToDevice);
  hipMemcpy(d_v, h_v, n * sizeof(float4), hipMemcpyHostToDevice);
  
  TIMERSTART(simulation)
  for (float t = 0; t < 0.1; t+= dt) {
    ComputeInteractions<<<blocks, threadsPerBlock>>>(n, d_p, d_v, dt );
    UpdatePosition<<<blocks, threadsPerBlock>>>(d_p, d_v, dt);
  }
  hipDeviceSynchronize();
  TIMERSTOP(simulation)

  hipMemcpy(h_p, d_p, n * sizeof(float4), hipMemcpyDeviceToHost);
  
  hipMemcpy(h_v, d_v, n * sizeof(float4), hipMemcpyDeviceToHost);
  TIMERSTOP(total)

  float ek = Ek(n, h_v);
  float ep = Ep(n, h_p);
  std::cout << "Etot: " <<ek+ep <<std::endl;
  
  // Free pinned memory
  hipHostFree(h_p);
  hipHostFree(h_v);

  // Free memory on device
  hipFree(d_p);
  hipFree(d_v);

  return 0;
}