#include "hip/hip_runtime.h"
#include "utilities/time_utils.h"
#include "utilities/nbody_helpers.h"
#include <iostream>
#include <algorithm>
#include <omp.h>

#define MAX_THREADS_PER_BLOCK 1024

void setDeviceByName(const char* deviceName) {
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  int targetDevice = -1;
  for (int i = 0; i < deviceCount; ++i) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, i);
    std::string fullName(deviceProp.name);
    std::string shortName(deviceName);
    // Convert both names to lowercase for case-insensitive comparison
    std::transform(fullName.begin(), fullName.end(), fullName.begin(), ::tolower);
    std::transform(shortName.begin(), shortName.end(), shortName.begin(), ::tolower);
    if (fullName.find(shortName) != std::string::npos) {
      targetDevice = i;
      break;
    }
  }
  if (targetDevice != -1) {
    hipSetDevice(targetDevice);
    std::cout << "Device " << deviceName << " set as the current device." << std::endl;
  } else {
    std::cerr << "Device " << deviceName << " not found." << std::endl;
    exit(1);
  }
}

float inline xoxyi_rand(unsigned int *seed){
  return (float)rand_r(seed) / (float) RAND_MAX;
}

float inline Ep(const int n, float4 *p) {
  float Epot = 0.0f;
#pragma omp parallel for reduction(+:Epot)
  for (int i = 0; i < n; ++i) {
    for (int j = i + 1; j < n; ++j) {
      float D, x, y, z;
      x = p[i].x - p[j].x;
      y = p[i].y - p[j].y;
      z = p[i].z - p[j].z;
      D = sqrtf(x * x + y * y + z * z);
      Epot += -1.0f * p[i].w * p[j].w / D;
    }
  }
  return Epot;
}

float inline Ek(const int n, float4 *v) {
  float Ekin = 0.0;
#pragma omp parallel for reduction(+:Ekin)
  for (int i = 0; i < n; ++i) {
    Ekin += 0.5f * v[i].w * (v[i].x * v[i].x + v[i].y * v[i].y + v[i].z * v[i].z);
  }
  return Ekin;
}

static inline void scale4DArray(const int n, float4 *m, const float scale) {
#pragma omp parallel for
  for (int i = 0; i < n; ++i) {
    m[i].x *= scale;
    m[i].y *= scale;
    m[i].z *= scale;
  }
}

void InitPosVel(const int n, float4 *p, float4 *v, int seed) {
#pragma omp parallel
 { 
  unsigned int seedT = omp_get_thread_num() + seed * omp_get_num_threads();	 
  #pragma omp for
    for (int i = 0; i < n; ++i) {
      float R, X, Y;
      float mi = _M / n;
      R = xoxyi_rand(&seedT);
      X = acosf(1.0f - 2.0f * xoxyi_rand(&seedT));
      Y = xoxyi_rand(&seedT) * 2.0f * _PI;
      // https://www.researchgate.net/figure/Figure-A1-Spherical-coordinates_fig8_284609648
      p[i] = make_float4(R * sinf(X) * cosf(Y), R * sinf(X) * sinf(Y), R * cosf(X), mi);
      v[i] = make_float4(1.0f - 2.0f * xoxyi_rand(&seedT), 1.0f - 2.0f * xoxyi_rand(&seedT), 1.0f - 2.0f * xoxyi_rand(&seedT), mi);
    }
  }
}

void Move2Center(const int n, float4 *p, float4 *v) {
  float3 pp = make_float3(0.0f, 0.0f, 0.0f);
  float3 vv = make_float3(0.0f, 0.0f, 0.0f);
  float ppx = 0;
  float ppy = 0;
  float ppz = 0;
  float vvx = 0;
  float vvy = 0;
  float vvz = 0;
  int i;
#pragma omp parallel for reduction(+:ppx,ppy,ppz,vvx,vvy,vvz)
  for (i = 0; i < n; ++i) {
    ppx += p[i].x * p[i].w;
    ppy += p[i].y * p[i].w;
    ppz += p[i].z * p[i].w;

    vvx += v[i].x * v[i].w;
    vvy += v[i].y * v[i].w;
    vvz += v[i].z * v[i].w;
  }

  pp.x = ppx;
  pp.y = ppy;
  pp.z = ppz;

  vv.x = vvx;
  vv.y = vvy;
  vv.z = vvz;

  pp.x /= _M;
  pp.y /= _M;
  pp.z /= _M;
  vv.x /= _M;
  vv.y /= _M;
  vv.z /= _M;

#pragma omp parallel for
  for (i = 0; i < n; ++i) {
    p[i].x -= pp.x;
    p[i].y -= pp.y;
    p[i].z -= pp.z;
    v[i].x -= vv.x;
    v[i].y -= vv.y;
    v[i].z -= vv.z;
  }
}

void RescaleEnergy(const int n, float4 *p, float4 *v) {
  // Aarseth, 2003, Algorithm 7.2.
  float Epot = Ep(n, p);
  float Ekin = Ek(n, v);
  float virialRatio = 0.5f;
  float Qv = sqrtf(virialRatio * fabsf(Epot) / Ekin);
  scale4DArray(n, v, Qv);
  float beta = fabsf((1 - virialRatio) * Epot / (Epot + Ekin));

  scale4DArray(n, p, beta);
  scale4DArray(n, v, 1.0f / (sqrtf(beta)));

  // After first scale Ekin is -0.5Epot but E0 != -0.25.
  // So just scale up or down as needed.
  Epot = Ep(n, p);
  beta = Epot / -0.5f;
  scale4DArray(n, p, beta);
  scale4DArray(n, v, 1.0f / sqrtf(beta));
}

void InitBodies(const int n, float4 *p, float4 *v, int seed = 0) {
  // Initialize masses equally
  InitPosVel(n, p, v, seed);

  // Translate bodies to move the center of mass on center of the coordinate
  // system
  Move2Center(n, p, v);

  // Rescale energy
  RescaleEnergy(n, p, v);
}

__global__ void ComputeInteractions(const int n, float4 *p, float4 *v, const float dt) {
  float fx = 0.0f;
  float fy = 0.0f;
  float fz = 0.0f;
  auto i = blockDim.x * blockIdx.x + threadIdx.x;
  #pragma unroll
  for (int j = 0; j < n; ++j) {
      // compute distance pair
      auto dx = p[j].x - p[i].x;
      auto dy = p[j].y - p[i].y;
      auto dz = p[j].z - p[i].z;

      auto d = dx * dx + dy * dy + dz * dz + _SOFTENING*_SOFTENING;
      auto d_inv = 1.0f / sqrtf(d);
      auto d_inv3 = d_inv * d_inv * d_inv;

      fx += d_inv3 * p[j].w * dx;
      fy += d_inv3 * p[j].w * dy;
      fz += d_inv3 * p[j].w * dz;  
  }
  v[i].x += fx * dt;
  v[i].y += fy * dt;
  v[i].z += fz * dt;
}

__global__ void UpdatePosition(float4 *p, float4 *v, const float dt) {
  auto i = blockDim.x * blockIdx.x + threadIdx.x;
  p[i].x += v[i].x * dt;
  p[i].y += v[i].y * dt;
  p[i].z += v[i].z * dt;
}


int main (int argc, char **argv) {
  int seed = 0;
  if (argc < 2) {
    std::cerr << "Must specify the number of bodies" << std::endl;
    exit(1);
  }
  if (argc == 3)
    seed = atoi(argv[2]);
  else  
    srand(seed);

  int n = atoi(argv[1]);
  const float dt = 0.01f; 
  
  float4 *h_p, *h_v;

  setDeviceByName(argv[3]);

  // Allocate pinned memory
  hipHostMalloc(&h_p, n * sizeof(float4));
  hipHostMalloc(&h_v, n * sizeof(float4));

  // Init Bodies
  TIMERSTART(init)
  InitBodies(n, h_p, h_v, seed);
#ifdef MONITOR_ENERGY
  {
  float ek = Ek(n, h_v);
  float ep = Ep(n, h_p);
  std::cout << "Etot: " <<ek+ep <<std::endl;
  }
#endif
  TIMERSTOP(init)

  // Allocate memory on the device
  float4 *d_p, *d_v;
  hipMalloc(&d_p, n * sizeof(float4));
  hipMalloc(&d_v, n * sizeof(float4));

  dim3 blocks, threadsPerBlock;
  if (n < MAX_THREADS_PER_BLOCK) {
    blocks = dim3(1);
    threadsPerBlock = dim3(n);
  } else {
    // assuming that n is a power of two
    blocks = dim3(n / MAX_THREADS_PER_BLOCK);
    threadsPerBlock = dim3(MAX_THREADS_PER_BLOCK);
  }

  TIMERSTART(total)
  hipMemcpy(d_p, h_p, n * sizeof(float4), hipMemcpyHostToDevice);
  hipMemcpy(d_v, h_v, n * sizeof(float4), hipMemcpyHostToDevice);
  
  TIMERSTART(simulation)
  for (float t = 0; t < 100; t+= dt) {
    ComputeInteractions<<<blocks, threadsPerBlock>>>(n, d_p, d_v, dt );
    UpdatePosition<<<blocks, threadsPerBlock>>>(d_p, d_v, dt);
    hipMemcpy(h_p, d_p, n * sizeof(float4), hipMemcpyDeviceToHost);
    hipMemcpy(h_v, d_v, n * sizeof(float4), hipMemcpyDeviceToHost);
#ifdef MONITOR_ENERGY
    {
      float ek = Ek(n, h_v);
      float ep = Ep(n, h_p);
      std::cout << "Etot: " <<ek+ep <<std::endl;
    }
#endif
  }
  hipDeviceSynchronize();
  TIMERSTOP(simulation)

  //hipMemcpy(h_p, d_p, n * sizeof(float4), hipMemcpyDeviceToHost);
  //hipMemcpy(h_v, d_v, n * sizeof(float4), hipMemcpyDeviceToHost);
  TIMERSTOP(total)
#ifdef MONITOR_ENERGY
  {
  float ek = Ek(n, h_v);
  float ep = Ep(n, h_p);
  std::cout << "Etot: " <<ek+ep <<std::endl;
  }
#endif
  // Free pinned memory
  hipHostFree(h_p);
  hipHostFree(h_v);

  // Free memory on device
  hipFree(d_p);
  hipFree(d_v);

  return 0;
}
